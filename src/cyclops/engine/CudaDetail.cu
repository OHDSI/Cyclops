#define THRUST_IGNORE_CUB_VERSION_CHECK

#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/fill.h>

#include "CudaDetail.h"

template <typename RealType>
void resizeAndCopyToDeviceCuda(const std::vector<RealType>& hostVec, thrust::device_vector<RealType>& deviceVec, hipStream_t* stream)
{
	deviceVec.resize(hostVec.size());
	hipMemcpyAsync(thrust::raw_pointer_cast(deviceVec.data()),
			thrust::raw_pointer_cast(hostVec.data()),
			deviceVec.size()*sizeof(RealType),
			hipMemcpyHostToDevice, stream[0]);
	hipStreamSynchronize(stream[0]);
}

template void resizeAndCopyToDeviceCuda<double>(const std::vector<double>& hostVec, thrust::device_vector<double>& deviceVec, hipStream_t* stream);
template void resizeAndCopyToDeviceCuda<float>(const std::vector<float>& hostVec, thrust::device_vector<float>& deviceVec, hipStream_t* stream);
template void resizeAndCopyToDeviceCuda<int>(const std::vector<int>& hostVec, thrust::device_vector<int>& deviceVec, hipStream_t* stream);
template void resizeAndCopyToDeviceCuda<unsigned int>(const std::vector<unsigned int>& hostVec, thrust::device_vector<unsigned int>& deviceVec, hipStream_t* stream);

